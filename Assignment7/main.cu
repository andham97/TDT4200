#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
extern "C" {
    #include "libs/bitmap.h"
}

#define ERROR_EXIT -1
#define BLOCKX 8
#define BLOCKY 8
#define PIXEL(i,j,w) ((i)+(j)*(w))

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
 * Get system time to microsecond precision (ostensibly, the same as MPI_Wtime),
 * returns time in seconds
 */
double walltime ( void ) {
	static struct timeval t;
	gettimeofday ( &t, NULL );
	return ( t.tv_sec + 1e-6 * t.tv_usec );
}

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5
// If you apply another filter, remember not only to exchange
// the filter but also the filterFactor and the correct dimension.
/*
int const sobelYFilter[] = {-1, -2, -1,
                             0,  0,  0,
                             1,  2,  1};
float const sobelYFilterFactor = (float) 1.0;

int const sobelXFilter[] = {-1, -0, -1,
                            -2,  0, -2,
                            -1,  0, -1 , 0};
float const sobelXFilterFactor = (float) 1.0;

*/
int const laplacian1Filter[] = {  -1,  -4,  -1,
                                  -4,  20,  -4,
                                  -1,  -4,  -1};

float const laplacian1FilterFactor = (float) 1.0;/*

int const laplacian2Filter[] = { 0,  1,  0,
                                 1, -4,  1,
                                 0,  1,  0};
float const laplacian2FilterFactor = (float) 1.0;

int const laplacian3Filter[] = { -1,  -1,  -1,
                                  -1,   8,  -1,
                                  -1,  -1,  -1};
float const laplacian3FilterFactor = (float) 1.0;


//Bonus Filter:

int const gaussianFilter[] = { 1,  4,  6,  4, 1,
                               4, 16, 24, 16, 4,
                               6, 24, 36, 24, 6,
                               4, 16, 24, 16, 4,
                               1,  4,  6,  4, 1 };

float const gaussianFilterFactor = (float) 1.0 / 256.0;*/

__global__ void deviceApplyFilter(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  unsigned int y = blockIdx.y * BLOCKY + threadIdx.y;
  unsigned int x = blockIdx.x * BLOCKX + threadIdx.x;
  if (y >= height || x >= width) {
    return;
  }
  int aggregate = 0;
  for (unsigned int ky = 0; ky < filterDim; ky++) {
    int nky = filterDim - 1 - ky;
    for (unsigned int kx = 0; kx < filterDim; kx++) {
      int nkx = filterDim - 1 - kx;

      int yy = y + (ky - filterCenter);
      int xx = x + (kx - filterCenter);
      if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
        aggregate += in[PIXEL(xx, yy, width)] * filter[nky * filterDim + nkx];
    }
  }
  aggregate *= filterFactor;
  if (aggregate > 0) {
    out[PIXEL(x, y, width)] = (aggregate > 255) ? 255 : aggregate;
  } else {
    out[PIXEL(x, y, width)] = 0;
  }
}


// Apply convolutional filter on image data
void applyFilter(unsigned char **out, unsigned char **in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++) {
    for (unsigned int x = 0; x < width; x++) {
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[yy][xx] * filter[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[y][x] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[y][x] = 0;
      }
    }
  }
}


void help(char const *exec, char const opt, char const *optarg) {
    FILE *out = stdout;
    if (opt != 0) {
        out = stderr;
        if (optarg) {
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        } else {
            fprintf(out, "Invalid parameter - %c\n", opt);
        }
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

    fprintf(out, "\n");
    fprintf(out, "Example: %s in.bmp out.bmp -i 10000\n", exec);
}

int main(int argc, char **argv) {
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  int ret = 0;

  static struct option const long_options[] =  {
      {"help",       no_argument,       0, 'h'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}
  };

  static char const * short_options = "hi:";
  {
    char *endptr;
    int c;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1) {
      switch (c) {
      case 'h':
        help(argv[0],0, NULL);
        return 0;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg) {
          help(argv[0], c, optarg);
          return ERROR_EXIT;
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind+1)) {
    help(argv[0],' ',"Not enough arugments");
    return ERROR_EXIT;
  }
  input = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(input, argv[optind], strlen(argv[optind]));
  optind++;

  output = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(output, argv[optind], strlen(argv[optind]));
  optind++;

  /*
    End of Parameter parsing!
   */

  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0,0);
  if (image == NULL) {
    fprintf(stderr, "Could not allocate new image!\n");
  }

  if (loadBmpImage(image, input) != 0) {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    return ERROR_EXIT;
  }


  // Create a single color channel image. It is easier to work just with one color
  bmpImageChannel *imageChannel = newBmpImageChannel(image->width, image->height);
  if (imageChannel == NULL) {
    fprintf(stderr, "Could not allocate new image channel!\n");
    freeBmpImage(image);
    return ERROR_EXIT;
  }

  // Extract from the loaded image an average over all colors - nothing else than
  // a black and white representation
  // extractImageChannel and mapImageChannel need the images to be in the exact
  // same dimensions!
  // Other prepared extraction functions are extractRed, extractGreen, extractBlue
  if(extractImageChannel(imageChannel, image, extractAverage) != 0) {
    fprintf(stderr, "Could not extract image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }

  // Pointers to GPU data locations
  unsigned char *processChannel;
  unsigned char *resultChannel;
  int *filter;

  // Initialize GPU data locations for image data
  cudaErrorCheck(hipMalloc((void **) &processChannel, imageChannel->width * imageChannel->height));
  cudaErrorCheck(hipMalloc((void **) &resultChannel, imageChannel->width * imageChannel->height));
  cudaErrorCheck(hipMalloc((void **) &filter, 9 * sizeof(int)));

  // Copy original image data to GPU memory
  cudaErrorCheck(hipMemcpy(resultChannel, imageChannel->rawdata, imageChannel->width * imageChannel->height, hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpy(filter, laplacian1Filter, 9 * sizeof(int), hipMemcpyHostToDevice));

  //Here we do the actual computation!
  // imageChannel->data is a 2-dimensional array of unsigned char which is accessed row first ([y][x])
  bmpImageChannel *processImageChannel = newBmpImageChannel(imageChannel->width, imageChannel->height);

  // CPU Processing
  double start = walltime();
  for (unsigned int i = 0; i < iterations; i ++) {
    applyFilter(processImageChannel->data,
                imageChannel->data,
                imageChannel->width,
                imageChannel->height,
                (int *)laplacian1Filter, 3, laplacian1FilterFactor);
    //Swap the data pointers
    unsigned char ** tmp = processImageChannel->data;
    processImageChannel->data = imageChannel->data;
    imageChannel->data = tmp;
    unsigned char * tmp_raw = processImageChannel->rawdata;
    processImageChannel->rawdata = imageChannel->rawdata;
    imageChannel->rawdata = tmp_raw;
  }
  double end = walltime();
  printf("CPU Time: %f", (end - start));

  // GPU processing
  start = walltime();
  for (unsigned int i = 0; i < iterations; i ++) {
    dim3 gridBlock(imageChannel->width / BLOCKX, imageChannel->height / BLOCKY);
    dim3 threadBlock(BLOCKX, BLOCKY);
    deviceApplyFilter<<<gridBlock, threadBlock>>>(processChannel, resultChannel, imageChannel->width, imageChannel->height, filter, 3, laplacian1FilterFactor);
    cudaErrorCheck(hipGetLastError());
    unsigned char *t = processChannel;
    processChannel = resultChannel;
    resultChannel = t;
  }
  end = walltime();
  printf("GPU Time: %f", (end - start));
  
  freeBmpImageChannel(processImageChannel);
  hipFree(processChannel);
  hipFree(filter);

  bmpImageChannel *gpuChannel = newBmpImageChannel(imageChannel->width, imageChannel->height);

  hipMemcpy(gpuChannel->rawdata, resultChannel, imageChannel->width * imageChannel->height, hipMemcpyDeviceToHost);

  int diff = 0;
  for (unsigned int i = 0; i < imageChannel->width * imageChannel->height; i++) {
    if (imageChannel->rawdata[i] != gpuChannel->rawdata[i])
      diff++;
  }

  printf("Diff is: %d/%d", diff, (imageChannel->width * imageChannel->height));

  // Map our single color image back to a normal BMP image with 3 color channels
  // mapEqual puts the color value on all three channels the same way
  // other mapping functions are mapRed, mapGreen, mapBlue
  if (mapImageChannel(image, imageChannel, mapEqual) != 0) {
    fprintf(stderr, "Could not map image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }
  freeBmpImageChannel(imageChannel);
  hipFree(resultChannel);

  //Write the image back to disk
  if (saveBmpImage(image, output) != 0) {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    return ERROR_EXIT;
  };

  ret = 0;
  if (input)
    free(input);
  if (output)
    free(output);
  return ret;
};
